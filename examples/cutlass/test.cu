#include "hip/hip_runtime.h"
#include "cutlass/gemm/device/gemm.h"

using ColumnMajor = cutlass::layout::ColumnMajor;

using CutlassGemm = cutlass::gemm::device::Gemm<float,        // Data-type of A matrix
                                                ColumnMajor,  // Layout of A matrix
                                                float,        // Data-type of B matrix
                                                ColumnMajor,  // Layout of B matrix
                                                float,        // Data-type of C matrix
                                                ColumnMajor>; // Layout of C matrix

// This is a template kernel
extern "C"
template __global__ void cutlass::Kernel<CutlassGemm::GemmKernel>(CutlassGemm::GemmKernel::Params);

// This is a normal kernel
extern "C"
__global__ void InitializeMatrix_kernel(
  float *matrix,
  int rows,
  int columns,
  int seed = 0) {

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;

  if (i < rows && j < columns) {
    int offset = i + j * rows;

    // Generate arbitrary elements.
    int const k = 16807;
    int const m = 16;
    float value = float(((offset + seed) * k % m) - m / 2);

    matrix[offset] = value;
  }
}
