#include "hip/hip_runtime.h"
#include "layer.h"
#include "util.h"
#include <iostream>
#include <fstream>
#include <string>
#include <exception>

// Constructor
Layer::Layer(int M, int N, int O)
{
	this->M = M;
	this->N = N;
	this->O = O;

	float h_bias[N];
	float h_weight[N][M];

	output = NULL;
	preact = NULL;
	bias   = NULL;
	weight = NULL;

	for (int i = 0; i < N; ++i) {
		h_bias[i] = 0.5f - float(rand()) / float(RAND_MAX);

		for (int j = 0; j < M; ++j) {
			h_weight[i][j] = 0.5f - float(rand()) / float(RAND_MAX);
		}
	}

	hipMalloc(&output, sizeof(float) * O);
	hipMalloc(&preact, sizeof(float) * O);

	hipMalloc(&bias, sizeof(float) * N);

	hipMalloc(&weight, sizeof(float) * M * N);

	hipMalloc(&d_output, sizeof(float) * O);
	hipMalloc(&d_preact, sizeof(float) * O);
	hipMalloc(&d_weight, sizeof(float) * M * N);

	hipMemcpy(bias, h_bias, sizeof(float) * N, hipMemcpyHostToDevice);

	hipMemcpy(weight, h_weight, sizeof(float) * M * N, hipMemcpyHostToDevice);
}

// Load from a previously saved layer. Path should point to a file as saved by
// Layer::export(). If enableTrain is true, memory will be allocated for
// training purposes, otherwise the layer is loaded for inference only and can
// only be used for forward_pass().
Layer::Layer(std::string path, bool enableTrain)
{
  std::ifstream inF(path, std::ios::binary);
  if(inF.fail()) {
    throw std::runtime_error("Failed to open layer file\n");
  }

  inF.read((char*)&M, sizeof(int));
  inF.read((char*)&N, sizeof(int));
  inF.read((char*)&O, sizeof(int));
  if(inF.fail()) {
    throw std::runtime_error("Failed to read layer file\n");
  }

  float h_bias[N];
  float h_weight[N][M];

  inF.read((char*)h_bias, sizeof(float) * N);
  inF.read((char*)h_weight, sizeof(float) * M * N);
  inF.close();

	hipMalloc(&output, sizeof(float) * O);
	hipMalloc(&preact, sizeof(float) * O);

	hipMalloc(&bias, sizeof(float) * N);
	hipMalloc(&weight, sizeof(float) * M * N);

	hipMemcpy(bias, h_bias, sizeof(float) * N, hipMemcpyHostToDevice);
	hipMemcpy(weight, h_weight, sizeof(float) * M * N, hipMemcpyHostToDevice);

  if(enableTrain) {
    hipMalloc(&d_output, sizeof(float) * O);
    hipMalloc(&d_preact, sizeof(float) * O);
    hipMalloc(&d_weight, sizeof(float) * M * N);
  }
}

// Destructor
Layer::~Layer()
{
	hipFree(output);
	hipFree(preact);

	hipFree(bias);

	hipFree(weight);

	hipFree(d_output);
	hipFree(d_preact);
	hipFree(d_weight);
}

// Save the layer to a file
bool Layer::save(std::string path) {
  std::ofstream outF(path, std::ios::binary | std::ios::trunc);
  if(outF.fail()) {
    printf("Failed to write layer to %s\n", path.c_str());
    return false;
  }

  float h_bias[N];
  float h_weight[N][M];

  hipMemcpy(h_bias, bias, sizeof(float) * N, hipMemcpyDeviceToHost);
  hipMemcpy(h_weight, weight, sizeof(float) * M * N, hipMemcpyDeviceToHost);

  outF.write((char*)&M, sizeof(int));
  outF.write((char*)&N, sizeof(int));
  outF.write((char*)&O, sizeof(int));
  outF.write((char*)h_bias, sizeof(float) * N);
  outF.write((char*)h_weight, sizeof(float) * M * N);
  outF.close();

  if(outF.fail()) {
    printf("Failed to write layer to %s\n", path.c_str());
    return false;
  }

  return true;
}

// Send data one row from dataset to the GPU
void Layer::setOutput(float *data)
{
	hipMemcpy(output, data, sizeof(float) * O, hipMemcpyHostToDevice);
}

// Reset GPU memory between iterations
void Layer::clear()
{
	hipMemset(output, 0x00, sizeof(float) * O);
	hipMemset(preact, 0x00, sizeof(float) * O);
}

void Layer::bp_clear()
{
	hipMemset(d_output, 0x00, sizeof(float) * O);
	hipMemset(d_preact, 0x00, sizeof(float) * O);
	hipMemset(d_weight, 0x00, sizeof(float) * M * N);
}

