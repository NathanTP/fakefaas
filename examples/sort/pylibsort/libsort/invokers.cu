// Various invocation methods for libsort
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <algorithm>
#include <atomic>

#include "sort.h"
#include "utils.h"

// Perform a partial sort of bits [offset, width). boundaries will contain the
// index of the first element of each unique group value (each unique value of
// width bits), it must be 2^width elements long.
extern "C" bool gpuPartial(uint32_t* h_in, uint32_t *boundaries, size_t h_in_len, uint32_t offset, uint32_t width) {
    //auto-releases the reservation (if any) on destruction
    auto ctx = std::make_unique<cudaReservation>();
    if(!ctx->reserveDevice()) {
      return false;
    }

    int v;
    auto res = hipDeviceGetAttribute(&v, hipDeviceAttributePciBusId, 0);
    if(res != hipSuccess) {
      fprintf(stderr, "Failed to get attr: %s\n", hipGetErrorString(res));
      return false;
    }

    //The sort internally only supports 32bit sizes
    if(h_in_len > UINT32_MAX) {
      fprintf(stderr, "Input array length must be less than 2^32\n");
      return false;
    }
    SortState state (h_in, h_in_len);

    state.Step(offset, width);
    state.GetResult(h_in);
    state.GetBoundaries(boundaries, offset, width);

    return true;
}

// Sort provided input (h_in) in-place using the GPU
// Returns success status
extern "C" bool providedGpu(unsigned int* h_in, size_t h_in_len)
{
    //auto-releases the reservation (if any) on destruction
    auto ctx = std::make_unique<cudaReservation>();
    if(!ctx->reserveDevice()) {
      return false;
    }

    //The sort internally only supports 32bit sizes
    if(h_in_len > UINT32_MAX) {
      fprintf(stderr, "Input array length must be less than 2^32\n");
      return false;
    }
    SortState state(h_in, h_in_len);

    state.Step(0, 32);
    state.GetResult(h_in);

    return true;
}

// Sort provided input (in) using the CPU
// returns success status
extern "C" bool providedCpu(unsigned int* in, size_t len) {
    std::sort(in, in + len);
    return true;
}

extern "C" bool gpuPartialProfile(uint32_t* h_in, uint32_t *boundaries, size_t h_in_len, uint32_t offset, uint32_t width) {
  hipProfilerStart();
  auto ret = gpuPartial(h_in, boundaries, h_in_len, offset, width);
  hipProfilerStop();
  return ret;
}

extern "C" bool providedGpuProfile(unsigned int* h_in, size_t h_in_len) {
  hipProfilerStart();
  auto ret = providedGpu(h_in, h_in_len);
  hipProfilerStop();
  return ret;
}
