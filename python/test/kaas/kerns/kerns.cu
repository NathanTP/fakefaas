
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>

extern "C"
__global__ void verifyAndCopy(int len, int expect, int* in, int* out)
{
    int idx = blockIdx.x *blockDim.x + threadIdx.x;
    int val = in[idx];
    if(val != expect) {
        printf("Expected %d, got %d\n", expect, val);
        assert(val == expect);
    }
    out[idx] = -val;
}

extern "C"
__global__ void verifyArr(int len, int expect, int* in)
{
    int idx = blockIdx.x *blockDim.x + threadIdx.x;
    int val = in[idx];
    if(val != expect) {
        printf("(%d) Expected %d, got %d\n", idx, expect, val);
        assert(val == expect);
    }
}

extern "C"
__global__ void doublifyKern(float* a)
{
    int idx = threadIdx.x + blockIdx.x*4;
    a[idx] *= 2;
}

extern "C"
__global__ void sumKern(uint32_t *input, uint32_t *out)
{
    const int tid = threadIdx.x;

    auto step_size = 1;
    int number_of_threads = blockDim.x;

    while (number_of_threads > 0)
    {
        if (tid < number_of_threads)
        {
            const auto fst = tid * step_size * 2;
            const auto snd = fst + step_size;
            input[fst] += input[snd];
        }

        step_size <<= 1; 
        number_of_threads >>= 1;
        __syncthreads();
    }

    if(tid == 0) {
        *out = input[0];
    }
}

extern "C"
__global__ void prodKern(uint64_t len, uint32_t *v0, uint32_t *v1, uint32_t *vout)
{
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if (id < len) {
        vout[id] = v0[id] * v1[id];    
    }
}

extern "C"
__global__ void multipleOut(uint64_t len, uint32_t *inp, uint32_t *outIncremented, uint32_t *outDoubled)
{
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if (id < len) {
        outDoubled[id] = inp[id] * 2;
        outIncremented[id] = inp[id] + 1;
    }
}

// Row Major
#define flatIdx(R,C,NROW,NCOL) ((R*NCOL)+C)

// Generic matrix multiply.
// Original implementation by Aditi Singh (https://github.com/aditisingh/GPU-Gemm)
#define TILE_WIDTH 32
#define TILE_HEIGHT 32
extern "C"
__global__ void matmulKern(uint64_t *dims, float* outArr, float* array1, float* array0)
{
    uint64_t rows0 = dims[0];
    uint64_t cols0 = dims[1];
    uint64_t rows1 = dims[2];
    uint64_t cols1 = dims[3];

    //shared memory takes one tile at a time
    __shared__ float S1[TILE_WIDTH][TILE_HEIGHT];
    __shared__ float S2[TILE_HEIGHT][TILE_WIDTH];

    //threads x and y index for the current block
    unsigned int tx=threadIdx.x;	
    unsigned int ty=threadIdx.y;

    //row value using x and y index of current thread (respectively)
    unsigned int c=blockIdx.x*blockDim.x + threadIdx.x;	
    unsigned int r=blockIdx.y*blockDim.y + threadIdx.y;

    //register to store multiplication result initialized to zero
    float val=0;

    //going over all tiles one by one, with each m
    for(int m=0; m<1+((rows1-1)/TILE_WIDTH); m++)
    {
        //x and y thread value for current tile
        int var1=m*TILE_WIDTH+tx;
        int var2=m*TILE_WIDTH+ty;

        //copying a tile from array0
        //if the value is associated to a valid matrix coordinate in array0
        //then store it to shared memory S1
        if (r < rows0 && var1 < rows1) {
            //storing a "valid" value from array to shared memory
            S1[ty][tx] = array0[flatIdx(r, var1, rows0, cols0)];
        } else {
            //storing zero, since there is no valid value
            S1[ty][tx]=0;					
        }
        __syncthreads();

        //copying a tile from array1
        //if value is associates to a valid matrix coordinate in array1 then
        //store it to shared memory S2
        if(c < cols1 && var2 < rows1) {
            S2[ty][tx] = array1[flatIdx(var2, c, rows1, cols1)];
        } else { 
            //storing zero, since no valid value
            S2[ty][tx]=0;
        }
        __syncthreads();

        //going over entire tile, ty row in S1 and tx column in S2
        for(int i=0; i<TILE_WIDTH;i++) {
            val+=S1[ty][i]*S2[i][tx];
        }
        __syncthreads();
    }

    //removing degenerate cases
    if(r < rows0 && c< cols1) {
        //saving multiplication result to global memory
        outArr[flatIdx(r, c, rows0, cols1)] = val;
    }
}
