
#include <hip/hip_runtime.h>

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 700)
#define __shfl_sync(mask, var, lane, width) \
        __shfl((var), (lane), (width))

#define __shfl_down_sync(mask, var, offset, width) \
        __shfl_down((var), (offset), (width))

#define __shfl_up_sync(mask, var, offset, width) \
        __shfl_up((var), (offset), (width))
#endif


#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void fused_nn_batch_flatten_kernel0(float* __restrict__ placeholder, float* __restrict__ tensor) {
  tensor[(((int)threadIdx.x))] = placeholder[(((int)threadIdx.x))];
}

extern "C" __global__ void fused_nn_dense_nn_bias_add_nn_relu_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ placeholder2, float* __restrict__ T_relu) {
  float T_dense_rf[1];
  __shared__ float red_buf0[64];
  __shared__ float T_dense[1];
  T_dense_rf[(0)] = 0.000000e+00f;
  for (int k_outer = 0; k_outer < 2; ++k_outer) {
    T_dense_rf[(0)] = (T_dense_rf[(0)] + (placeholder[(((k_outer * 64) + ((int)threadIdx.x)))] * placeholder1[((((((int)blockIdx.x) * 128) + (k_outer * 64)) + ((int)threadIdx.x)))]));
  }
  __syncthreads();
  ((volatile float*)red_buf0)[(((int)threadIdx.x))] = T_dense_rf[(0)];
  __syncthreads();
  if (((int)threadIdx.x) < 32) {
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 32))]);
  }
  __syncthreads();
  if (((int)threadIdx.x) < 16) {
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 16))]);
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 8))]);
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 4))]);
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 2))]);
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 1))]);
  }
  __syncthreads();
  if (((int)threadIdx.x) == 0) {
    T_dense[(0)] = ((volatile float*)red_buf0)[(0)];
  }
  if (((int)threadIdx.x) == 0) {
    T_relu[(((int)blockIdx.x))] = max((T_dense[(0)] + placeholder2[(((int)blockIdx.x))]), 0.000000e+00f);
  }
}

extern "C" __global__ void fused_nn_softmax_kernel0(float* __restrict__ placeholder, float* __restrict__ T_softmax_norm) {
  float normal_reduce_temp0[1];
  float red_buf0[1];
  float T_softmax_exp[1];
  float normal_reduce_temp01[1];
  float red_buf01[1];
  normal_reduce_temp0[(0)] = -3.402823e+38f;
  if (((int)threadIdx.x) < 10) {
    normal_reduce_temp0[(0)] = max(normal_reduce_temp0[(0)], placeholder[(((int)threadIdx.x))]);
  }
  uint mask[1];
  float t0[1];
  red_buf0[(0)] = normal_reduce_temp0[(0)];
  mask[(0)] = __activemask();
  t0[(0)] = __shfl_down_sync(mask[(0)], red_buf0[(0)], 16, 32);
  red_buf0[(0)] = max(red_buf0[(0)], t0[(0)]);
  t0[(0)] = __shfl_down_sync(mask[(0)], red_buf0[(0)], 8, 32);
  red_buf0[(0)] = max(red_buf0[(0)], t0[(0)]);
  t0[(0)] = __shfl_down_sync(mask[(0)], red_buf0[(0)], 4, 32);
  red_buf0[(0)] = max(red_buf0[(0)], t0[(0)]);
  t0[(0)] = __shfl_down_sync(mask[(0)], red_buf0[(0)], 2, 32);
  red_buf0[(0)] = max(red_buf0[(0)], t0[(0)]);
  t0[(0)] = __shfl_down_sync(mask[(0)], red_buf0[(0)], 1, 32);
  red_buf0[(0)] = max(red_buf0[(0)], t0[(0)]);
  red_buf0[(0)] = __shfl_sync(mask[(0)], red_buf0[(0)], 0, 32);
  if (((int)threadIdx.x) < 10) {
    T_softmax_exp[(0)] = __expf((placeholder[(((int)threadIdx.x))] - red_buf0[(0)]));
  }
  normal_reduce_temp01[(0)] = 0.000000e+00f;
  if (((int)threadIdx.x) < 10) {
    normal_reduce_temp01[(0)] = (normal_reduce_temp01[(0)] + __shfl_sync(__activemask(), T_softmax_exp[(0)], ((int)threadIdx.x), 32));
  }
  uint mask1[1];
  float t01[1];
  red_buf01[(0)] = normal_reduce_temp01[(0)];
  mask1[(0)] = __activemask();
  t01[(0)] = __shfl_down_sync(mask1[(0)], red_buf01[(0)], 16, 32);
  red_buf01[(0)] = (red_buf01[(0)] + t01[(0)]);
  t01[(0)] = __shfl_down_sync(mask1[(0)], red_buf01[(0)], 8, 32);
  red_buf01[(0)] = (red_buf01[(0)] + t01[(0)]);
  t01[(0)] = __shfl_down_sync(mask1[(0)], red_buf01[(0)], 4, 32);
  red_buf01[(0)] = (red_buf01[(0)] + t01[(0)]);
  t01[(0)] = __shfl_down_sync(mask1[(0)], red_buf01[(0)], 2, 32);
  red_buf01[(0)] = (red_buf01[(0)] + t01[(0)]);
  t01[(0)] = __shfl_down_sync(mask1[(0)], red_buf01[(0)], 1, 32);
  red_buf01[(0)] = (red_buf01[(0)] + t01[(0)]);
  red_buf01[(0)] = __shfl_sync(mask1[(0)], red_buf01[(0)], 0, 32);
  if (((int)threadIdx.x) < 10) {
    T_softmax_norm[(((int)threadIdx.x))] = (__shfl_sync(__activemask(), T_softmax_exp[(0)], ((int)threadIdx.x), 32) / red_buf01[(0)]);
  }
}

extern "C" __global__ void fused_nn_dense_nn_bias_add_nn_relu_1_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ placeholder2, float* __restrict__ T_relu) {
  float T_dense_rf[1];
  __shared__ float red_buf0[64];
  __shared__ float T_dense[1];
  T_dense_rf[(0)] = 0.000000e+00f;
  for (int k_outer = 0; k_outer < 13; ++k_outer) {
    if (((k_outer * 64) + ((int)threadIdx.x)) < 784) {
      T_dense_rf[(0)] = (T_dense_rf[(0)] + (placeholder[(((k_outer * 64) + ((int)threadIdx.x)))] * placeholder1[((((((int)blockIdx.x) * 784) + (k_outer * 64)) + ((int)threadIdx.x)))]));
    }
  }
  __syncthreads();
  ((volatile float*)red_buf0)[(((int)threadIdx.x))] = T_dense_rf[(0)];
  __syncthreads();
  if (((int)threadIdx.x) < 32) {
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 32))]);
  }
  __syncthreads();
  if (((int)threadIdx.x) < 16) {
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 16))]);
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 8))]);
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 4))]);
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 2))]);
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 1))]);
  }
  __syncthreads();
  if (((int)threadIdx.x) == 0) {
    T_dense[(0)] = ((volatile float*)red_buf0)[(0)];
  }
  if (((int)threadIdx.x) == 0) {
    T_relu[(((int)blockIdx.x))] = max((T_dense[(0)] + placeholder2[(((int)blockIdx.x))]), 0.000000e+00f);
  }
}

extern "C" __global__ void fused_nn_dense_nn_bias_add_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ placeholder2, float* __restrict__ T_add) {
  float T_dense_rf[1];
  __shared__ float red_buf0[64];
  __shared__ float T_dense[1];
  T_dense_rf[(0)] = 0.000000e+00f;
  T_dense_rf[(0)] = (T_dense_rf[(0)] + (placeholder[(((int)threadIdx.x))] * placeholder1[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))]));
  __syncthreads();
  ((volatile float*)red_buf0)[(((int)threadIdx.x))] = T_dense_rf[(0)];
  __syncthreads();
  if (((int)threadIdx.x) < 32) {
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 32))]);
  }
  __syncthreads();
  if (((int)threadIdx.x) < 16) {
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 16))]);
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 8))]);
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 4))]);
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 2))]);
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 1))]);
  }
  __syncthreads();
  if (((int)threadIdx.x) == 0) {
    T_dense[(0)] = ((volatile float*)red_buf0)[(0)];
  }
  if (((int)threadIdx.x) == 0) {
    T_add[(((int)blockIdx.x))] = (T_dense[(0)] + placeholder2[(((int)blockIdx.x))]);
  }
}

