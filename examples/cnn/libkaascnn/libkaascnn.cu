#include <hip/hip_runtime.h>
#include <stdio.h>
#include "libkaascnn.h"

extern "C" bool initLibkaascnn(void)
{
	hipError_t err = hipInit(0);
	if (err != hipSuccess) {
		fprintf(stderr, "CUDA initialisation failed with error code - %d\n", err);
		return false;
	}

  return true;
}
